#include "hip/hip_runtime.h"
#include "alpha_tree.cuh"
#include "cc_labelling.cuh"
#include "cuda_error.cuh"
#include "graph_creation.cuh"

#include "image.hh"

#include <algorithm>
#include <cmath>

using namespace utils;

constexpr int connectivity = 4;

void debug_display(int nb_site, int* labels, int* residual_list, bool verbose = false)
{
    if (verbose)
    {
        for (int i = 0; i < nb_site; ++i)
        {
            std::cout << "SITE: " << i << ", LABEL: " << labels[i] << ", RESIDUAL: ";
            for (uint j = i * connectivity; j - i * connectivity < connectivity; ++j)
            {
                std::cout << residual_list[j] << " ";
            }
            std::cout << std::endl;
        }
    }

    std::vector<int> unique;
    for (int i = 0; i < nb_site; ++i)
        unique.push_back(labels[i]);
    auto ip = std::unique(unique.begin(), unique.end());
    unique.resize(std::distance(unique.begin(), ip));

    std::cout << "Number of flatzone / Number of pixels: " << (double)unique.size() << " / " << (double)nb_site << std::endl;
}

int main()
{
    // // Image loading

    //    auto image = RGBImage::load("../batiment.png");

    //    int nb_site = image->height * image->width;
    //
    //    // Memory allocation
    //
    //    hipError_t rc = hipSuccess;
    //
    //    int* m_nn_list;
    //    rc = hipMallocManaged(&m_nn_list, connectivity * nb_site * sizeof(int));
    //    if (rc)
    //        abortError("Fail M_NN_LIST allocation");
    //    rc = hipMemset(m_nn_list, -1, connectivity * nb_site * sizeof(int));
    //    if (rc)
    //        abortError("Fail M_NN_LIST memset");
    //
    //    int* m_labels;
    //    rc = hipMallocManaged(&m_labels, nb_site * sizeof(int));
    //    if (rc)
    //        abortError("Fail M_LABELS allocation");
    //
    //    int* m_residual_list;
    //    rc = hipMallocManaged(&m_residual_list, connectivity * nb_site * sizeof(int));
    //    if (rc)
    //        abortError("Fail M_RESIDUAL_LIST allocation");
    //    rc = hipMemset(m_residual_list, -1, connectivity * nb_site * sizeof(int));
    //    if (rc)
    //        abortError("Fail M_RESIDUAL_LIST memset");
    //
    //    // Kernel setup
    //
    //    int bsize = 32;
    //    int w = std::ceil((float)image->width / bsize);
    //    int h = std::ceil((float)image->height / bsize);
    //
    //    dim3 dimBlock(bsize, bsize);
    //    dim3 dimGrid(w, h);
    //
    //    // Kernel launch
    //
    //    // Graph creation
    //    {
    //        create_graph_4<<<dimGrid, dimBlock>>>(image->pixels, m_nn_list, image->height, image->width);
    //        hipDeviceSynchronize();
    //    }
    //
    //    // Flat zone labelization
    //    {
    //        initialization_step<<<dimGrid, dimBlock>>>(m_nn_list, m_residual_list, m_labels, image->height, image->width);
    //        hipDeviceSynchronize();
    //
    //        analysis_step<<<dimGrid, dimBlock>>>(m_labels, image->height, image->width);
    //        hipDeviceSynchronize();
    //
    //        reduction_step<<<dimGrid, dimBlock>>>(m_residual_list, m_labels, image->height, image->width);
    //        hipDeviceSynchronize();
    //
    //        analysis_step<<<dimGrid, dimBlock>>>(m_labels, image->height, image->width);
    //        hipDeviceSynchronize();
    //    }
    //
    //    if (hipPeekAtLastError())
    //        abortError("Computation Error");
    //
    //    // Image reconstruction
    //
    //    for (int j = 0; j < image->height; ++j)
    //    {
    //        for (int i = 0; i < image->width; ++i)
    //        {
    //            int site = i + j * image->width;
    //            image->pixels[site] = image->pixels[m_labels[site]];
    //        }
    //    }
    //
    //    // Validity checks
    //
    //    image->save("flatzone_labelling.png");
    //
    //    debug_display(nb_site, m_labels, m_residual_list);
    //
    //    // Free memory
    //
    //    hipFree(m_nn_list);
    //    hipFree(m_labels);
    //    hipFree(m_residual_list);

    RGBPixel image[6] = {
        {100, 100, 100}, {100, 100, 150}, {20, 45, 79}, {100, 100, 100}, {100, 100, 150}, {20, 45, 79}};
    int height = 6;
    int width = 1;

    hipError_t rc = hipSuccess;

    RGBPixel* m_image;
    rc = hipMallocManaged(&m_image, sizeof(image));
    if (rc)
        abortError("Fail M_IMAGE allocation");
    rc = hipMemcpy(m_image, image, sizeof(image), hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail M_IMAGE memcpy");

    int* m_parent;
    rc = hipMallocManaged(&m_parent, (2 * height * width - 1) * sizeof(int));
    if (rc)
        abortError("Fail M_IMAGE allocation");

    double* m_levels;
    rc = hipMallocManaged(&m_levels, (2 * height * width - 1) * sizeof(double));
    if (rc)
        abortError("Fail M_LEVELS allocation");

    int bsize = 32;
    int w = std::ceil((float)(width + bsize) / bsize);
    int h = std::ceil((float)height / bsize);

    dim3 dimBlock(bsize, bsize);
    dim3 dimGrid(w, h);

    init_parent<<<dimGrid, dimBlock>>>(m_parent, height, width);

    hipDeviceSynchronize();

    for (int i = 0; i < 2 * height * width - 1; ++i)
        std::cout << m_parent[i] << ", ";
    std::cout << std::endl;

    //    w = std::ceil((float)width / bsize);
    //    h = std::ceil((float)height / bsize);
    //
    //    dimBlock = dim3(bsize, bsize);
    //    dimGrid = dim3(w, h);
    //
    //    build_alpha_tree_col<32><<<dimGrid, dimBlock>>>(m_image, m_parent, m_levels, height, width);
    //
    //    hipDeviceSynchronize();
    //
    //    for (int i = 0; i < 2 * height * width - 1; ++i)
    //        std::cout << m_parent[i] << ", ";
    //    std::cout << std::endl;

    hipFree(m_image);
    hipFree(m_parent);
    hipFree(m_levels);

    return 0;
}