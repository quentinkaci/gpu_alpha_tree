#include "hip/hip_runtime.h"
#include "cc_labelling.cuh"
#include "image.hh"

#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace utils;

// 4 connectivity
constexpr int connectivity = 4;

[[gnu::noinline]] void _abortError(const char* msg, const char* fname, int line)
{
    hipError_t err = hipGetLastError();
    std::cerr << "Error: " << err << std::endl;
    std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

float gradient(RGBPixel src, RGBPixel dst)
{
    return sqrt(pow(dst.r - src.r, 2) + pow(dst.g - src.g, 2) + pow(dst.b - src.b, 2));
}

void add_neighbour(std::vector<int>& nn_list, int site, int nn)
{
    int i = 0;
    while (nn_list[connectivity * site + i] != -1)
        ++i;
    nn_list[connectivity * site + i] = nn;
}

std::vector<int> create_graph_4(std::shared_ptr<RGBImage> image)
{
    std::vector<int> nn_list(connectivity * image->width * image->height, -1);

    for (int j = 0; j < image->height; ++j)
    {
        for (int i = 0; i < image->width; ++i)
        {
            int src_pos = i + j * image->width;

            if (j != image->height - 1)
            {
                int dst_pos = i + (j + 1) * image->width;
                if (gradient(image->pixels[src_pos], image->pixels[dst_pos]) == 0.f)
                {
                    add_neighbour(nn_list, src_pos, dst_pos);
                    add_neighbour(nn_list, dst_pos, src_pos);
                }
            }

            if (i != image->width - 1)
            {
                int dst_pos = (i + 1) + j * image->width;
                if (gradient(image->pixels[src_pos], image->pixels[dst_pos]) == 0.f)
                {
                    add_neighbour(nn_list, src_pos, dst_pos);
                    add_neighbour(nn_list, dst_pos, src_pos);
                }
            }
        }
    }

    return nn_list;
}

void debug_display(int nb_site, int* labels, int* residual_list, bool verbose = false)
{
    if (verbose)
    {
        for (int i = 0; i < nb_site; ++i)
        {
            std::cout << "SITE: " << i << ", LABEL: " << labels[i] << ", RESIDUAL: ";
            for (uint j = i * connectivity; j - i * connectivity < connectivity; ++j)
            {
                std::cout << residual_list[j] << " ";
            }
            std::cout << std::endl;
        }
    }

    std::vector<int> unique;
    for (int i = 0; i < nb_site; ++i)
        unique.push_back(labels[i]);
    auto ip = std::unique(unique.begin(), unique.end());
    unique.resize(std::distance(unique.begin(), ip));

    std::cout << "Number of flatzone / Number of pixels: " << (double)unique.size() << " / " << (double)nb_site << std::endl;
}

int main()
{
    auto image = RGBImage::load("../batiment.png");

    int nb_site = image->height * image->width;

    auto nn_list_vector = create_graph_4(image);
    auto nn_list = nn_list_vector.data();

    hipError_t rc = hipSuccess;

    int* m_nn_list;
    rc = hipMallocManaged(&m_nn_list, connectivity * nb_site * sizeof(int));
    if (rc)
        abortError("Fail M_NN_LIST allocation");
    hipMemcpy(m_nn_list, nn_list, connectivity * nb_site * sizeof(int), hipMemcpyHostToHost);

    int* m_labels;
    rc = hipMallocManaged(&m_labels, nb_site * sizeof(int));
    if (rc)
        abortError("Fail M_LABELS allocation");

    int* m_residual_list;
    rc = hipMallocManaged(&m_residual_list, connectivity * nb_site * sizeof(int));
    if (rc)
        abortError("Fail M_RESIDUAL_LIST allocation");
    hipMemset(m_residual_list, -1, connectivity * nb_site * sizeof(int));

    int bsize = 32;
    int w = std::ceil((float)image->width / bsize);
    int h = std::ceil((float)image->height / bsize);

    dim3 dimBlock(bsize, bsize);
    dim3 dimGrid(w, h);

    initialization_step<<<dimGrid, dimBlock>>>(m_nn_list, connectivity, m_residual_list, m_labels, image->height, image->width);
    hipDeviceSynchronize();

    analysis_step<<<dimGrid, dimBlock>>>(m_labels, image->height, image->width);
    hipDeviceSynchronize();

    reduction_step<<<dimGrid, dimBlock>>>(m_residual_list, connectivity, m_labels, image->height, image->width);
    hipDeviceSynchronize();

    analysis_step<<<dimGrid, dimBlock>>>(m_labels, image->height, image->width);
    hipDeviceSynchronize();

    for (int j = 0; j < image->height; ++j)
    {
        for (int i = 0; i < image->width; ++i)
        {
            int site = i + j * image->width;
            image->pixels[site] = image->pixels[m_labels[site]];
        }
    }

    image->save("flatzone_labelling.png");

    debug_display(nb_site, m_labels, m_residual_list);

    return 0;
}