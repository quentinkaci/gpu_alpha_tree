#include "hip/hip_runtime.h"
#include "cc_labelling.cuh"
#include "image.hh"

#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace utils;

// 4 connectivity
constexpr int connectivity = 4;

[[gnu::noinline]] void _abortError(const char* msg, const char* fname, int line)
{
    hipError_t err = hipGetLastError();
    std::cerr << "Error: " << err << std::endl;
    std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

float gradient(RGBPixel src, RGBPixel dst)
{
    return sqrt(pow(dst.r - src.r, 2) + pow(dst.g - src.g, 2) + pow(dst.b - src.b, 2));
}

void add_neighbour(std::vector<int>& nn_list, int site, int nn)
{
    int i = 0;
    while (nn_list[connectivity * site + i] != -1)
        ++i;
    nn_list[connectivity * site + i] = nn;
}

std::vector<int> create_graph_4(std::shared_ptr<RGBImage> image)
{
    std::vector<int> nn_list(connectivity * image->width * image->height, -1);

    for (int j = 0; j < image->height; ++j)
    {
        for (int i = 0; i < image->width; ++i)
        {
            int src_pos = i + j * image->width;

            if (j != image->height - 1)
            {
                int dst_pos = i + (j + 1) * image->width;
                if (gradient(image->pixels[src_pos], image->pixels[dst_pos]) == 0.f)
                {
                    add_neighbour(nn_list, src_pos, dst_pos);
                    add_neighbour(nn_list, dst_pos, src_pos);
                }
            }

            if (i != image->width - 1)
            {
                int dst_pos = (i + 1) + j * image->width;
                if (gradient(image->pixels[src_pos], image->pixels[dst_pos]) == 0.f)
                {
                    add_neighbour(nn_list, src_pos, dst_pos);
                    add_neighbour(nn_list, dst_pos, src_pos);
                }
            }
        }
    }

    return nn_list;
}

int main()
{
    // clang-format off
    //    int nn_list[] = {
    //        3, 10, -1, -1,
    //        4, 5, 15, -1,
    //        17, 18, -1, -1,
    //        0, 8, 9, 21,
    //        1, 11, -1, -1,
    //        1, 13, 20, -1,
    //        8, 9, -1, -1,
    //        20, -1, -1, -1,
    //        3, 6, -1, -1,
    //        3, 6, 10, -1,
    //        0, 9, -1, -1,
    //        4, 20, -1, -1,
    //        16, 19, 21, -1,
    //        5, 15, -1, -1,
    //        19, 21, -1, -1,
    //        1, 13, -1, -1,
    //        12, 21, -1, -1,
    //        2, 18, -1, -1,
    //        2, 17, -1, -1,
    //        12, 14, -1, -1,
    //        5, 7, 11, -1,
    //        3, 12, 14, 16
    //    };
    // clang-format on

    auto image = RGBImage::load("../batiment.png");

    auto nn_list_vect = create_graph_4(image);
    int nb_site = nn_list_vect.size() / connectivity;
    auto nn_list = nn_list_vect.data();

    hipError_t rc = hipSuccess;

    int* m_nn_list;
    rc = hipMallocManaged(&m_nn_list, connectivity * nb_site * sizeof(int));
    if (rc)
        abortError("Fail M_NN_LIST allocation");

    hipMemcpy(m_nn_list, nn_list, connectivity * nb_site * sizeof(int), hipMemcpyHostToHost);

    int* m_labels;
    rc = hipMallocManaged(&m_labels, nb_site * sizeof(int));
    if (rc)
        abortError("Fail M_LABELS allocation");

    int* m_residual_list;
    rc = hipMallocManaged(&m_residual_list, connectivity * nb_site * sizeof(int));
    if (rc)
        abortError("Fail M_RESIDUAL_LIST allocation");
    hipMemset(m_residual_list, -1, connectivity * nb_site * sizeof(int));

    for (int i = 0; i < nb_site; ++i)
        initialization_step(m_nn_list, connectivity, m_residual_list, m_labels, i);

    for (int i = 0; i < nb_site; ++i)
        anylisis_step(m_labels, i);

    for (int i = 0; i < nb_site; ++i)
    {
        reduction_step<<<1, 1>>>(m_residual_list, connectivity, m_labels, i);
        hipDeviceSynchronize();
    }

    for (int i = 0; i < nb_site; ++i)
        anylisis_step(m_labels, i);

    for (int j = 0; j < image->height; ++j)
    {
        for (int i = 0; i < image->width; ++i)
        {
            int site = i + j * image->width;
            image->pixels[site] = image->pixels[m_labels[site]];
        }
    }

    //    for (int j = 0; j < 50; ++j)
    //    {
    //        for (int i = 0; i < 50; ++i)
    //        {
    //            int site = i + j * image->width;
    //            std::cout << m_labels[site] << " ";
    //        }
    //        std::cout << std::endl;
    //    }

    std::vector<int> unique;
    for (int i = 0; i < nb_site; ++i)
        unique.push_back(m_labels[i]);
    auto ip = std::unique(unique.begin(), unique.end());
    unique.resize(std::distance(unique.begin(), ip));

    std::cout
        << (double)unique.size() / (double)nb_site << std::endl;

    image->save("flatzone_labelling.png");

    //    for (int i = 0; i < nb_site; ++i)
    //    {
    //        std::cout << "SITE: " << i << ", LABEL: " << m_labels[i] << ", RESIDUAL: ";
    //        for (uint j = i * connectivity; j - i * connectivity < connectivity; ++j)
    //        {
    //            std::cout << m_residual_list[j] << " ";
    //        }
    //        std::cout << std::endl;
    //    }

    return 0;
}